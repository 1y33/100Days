#pragma once

#include <algorithm>
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#define CEIL_DIV(M, N) (((M) + (N) - 1) / (N))
const int K9_NUM_THREADS = 256;

template <const int BM, const int BN, const int BK, const int TM, const int TN>
__global__ void __launch_bounds__(K9_NUM_THREADS)
    sgemmAutotuned(int M, int N, int K, float alpha, float *A, float *B,
                   float beta, float *C)
{
    const uint cRow = blockIdx.y;
    const uint cCol = blockIdx.x;

    constexpr int WM = TM * 16;
    constexpr int WN = TN * 16;
    constexpr int WMITER = CEIL_DIV(BM, WM);
    constexpr int WNITER = CEIL_DIV(BN, WN);

    const int threadCol = threadIdx.x % (WN / TN);
    const int threadRow = threadIdx.x / (WN / TN);

    __shared__ float As[BM * BK];
    __shared__ float Bs[BK * BN];

    A += cRow * BM * K;
    B += cCol * BN;
    C += cRow * BM * N + cCol * BN;

    const uint innerRowA = threadIdx.x / (BK / 4);
    const uint innerColA = threadIdx.x % (BK / 4);
    constexpr uint rowStrideA = (K9_NUM_THREADS * 4) / BK;
    const uint innerRowB = threadIdx.x / (BN / 4);
    const uint innerColB = threadIdx.x % (BN / 4);
    constexpr uint rowStrideB = K9_NUM_THREADS / (BN / 4);

    float threadResults[WMITER * WNITER * TM * TN] = {0.0};
    float regM[TM] = {0.0};
    float regN[TN] = {0.0};

    for (uint bkIdx = 0; bkIdx < K; bkIdx += BK)
    {
        for (uint offset = 0; offset + rowStrideA <= BM; offset += rowStrideA)
        {
            float4 tmp = reinterpret_cast<float4 *>(
                &A[(innerRowA + offset) * K + innerColA * 4])[0];
            As[(innerColA * 4 + 0) * BM + innerRowA + offset] = tmp.x;
            As[(innerColA * 4 + 1) * BM + innerRowA + offset] = tmp.y;
            As[(innerColA * 4 + 2) * BM + innerRowA + offset] = tmp.z;
            As[(innerColA * 4 + 3) * BM + innerRowA + offset] = tmp.w;
        }

        for (uint offset = 0; offset + rowStrideB <= BK; offset += rowStrideB)
        {
            reinterpret_cast<float4 *>(
                &Bs[(innerRowB + offset) * BN + innerColB * 4])[0] =
                reinterpret_cast<float4 *>(
                    &B[(innerRowB + offset) * N + innerColB * 4])[0];
        }
        __syncthreads();

        for (uint wmIdx = 0; wmIdx < WMITER; ++wmIdx)
        {
            for (uint wnIdx = 0; wnIdx < WNITER; ++wnIdx)
            {
                for (uint dotIdx = 0; dotIdx < BK; ++dotIdx)
                {
                    for (uint i = 0; i < TM; ++i)
                    {
                        regM[i] = As[dotIdx * BM + (wmIdx * WM) + threadRow * TM + i];
                    }
                    for (uint i = 0; i < TN; ++i)
                    {
                        regN[i] = Bs[dotIdx * BN + (wnIdx * WN) + threadCol * TN + i];
                    }
                    for (uint resIdxM = 0; resIdxM < TM; ++resIdxM)
                    {
                        for (uint resIdxN = 0; resIdxN < TN; ++resIdxN)
                        {
                            threadResults[(wmIdx * TM + resIdxM) * (WNITER * TN) +
                                          wnIdx * TN + resIdxN] +=
                                regM[resIdxM] * regN[resIdxN];
                        }
                    }
                }
            }
        }
        __syncthreads();
        A += BK;
        B += BK * N;
    }

    for (uint wmIdx = 0; wmIdx < WMITER; ++wmIdx)
    {
        for (uint wnIdx = 0; wnIdx < WNITER; ++wnIdx)
        {
            float *C_interim = C + (wmIdx * WM * N) + (wnIdx * WN);
            for (uint resIdxM = 0; resIdxM < TM; resIdxM += 1)
            {
                for (uint resIdxN = 0; resIdxN < TN; resIdxN += 4)
                {
                    float4 tmp = reinterpret_cast<float4 *>(
                        &C_interim[(threadRow * TM + resIdxM) * N + threadCol * TN +
                                   resIdxN])[0];
                    const int i = (wmIdx * TM + resIdxM) * (WNITER * TN) + wnIdx * TN + resIdxN;
                    tmp.x = alpha * threadResults[i + 0] + beta * tmp.x;
                    tmp.y = alpha * threadResults[i + 1] + beta * tmp.y;
                    tmp.z = alpha * threadResults[i + 2] + beta * tmp.z;
                    tmp.w = alpha * threadResults[i + 3] + beta * tmp.w;
                    reinterpret_cast<float4 *>(&C_interim[(threadRow * TM + resIdxM) * N + threadCol * TN + resIdxN])[0] = tmp;
                }
            }
        }
    }
}
